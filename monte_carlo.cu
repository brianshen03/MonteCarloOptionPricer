#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <cmath>
#include <random>
#include <chrono>
#include <algorithm>
#include "live_data.hpp"
#include <hiprand/hiprand_kernel.h>


//stock option parameters
// S = stock price
// X = strike price
// T = time to expiration in years
// r = risk-free interest rate
// sigma = volatility of the stock price
struct OptionGPU {
    double S, X, T, r, sigma;
};

struct config {
    std::string ticker;
    int num_simulations = 1000000; // default to 1 million simulations
};

struct optionPrices {
    double call_price;
    double put_price;
};

std::vector<optionParams> trades;

//helper function to calculate CDF
double phi(double x) {
    return 0.5 * std::erfc(-x/std::sqrt(2.0));
}

//calculate option price using Black-Scholes formula
optionPrices calc_call(const optionParams& params) {

    double d1 = (std::log(params.S/params.X) + (params.r + (params.sigma*params.sigma)/2) * params.T)/
                (params.sigma*std::sqrt(params.T));

    double d2 = (std::log(params.S/params.X) + (params.r - (params.sigma*params.sigma)/2) * params.T)/
             (params.sigma*std::sqrt(params.T));

             
    double CDF_d1_c = phi(d1);
    double CDF_d2_c = phi(d2);
    double CDF_d1_p = phi(-d1);
    double CDF_d2_p = phi(-d2);
    
    double call = params.S * CDF_d1_c - params.X * std::exp(-params.r * params.T) * CDF_d2_c;
    double put = params.X * std::exp(-params.r * params.T) * CDF_d2_p - params.S * CDF_d1_p;
    optionPrices option_price = {call, put};
    return option_price;

}

// Monte Carlo simulation to estimate the option price
//each cuda thread does a simulation of the option price
__global__ void monte_carlo_simulation(const OptionGPU* trades_pointer, optionPrices* trades_results, int num_simulations, unsigned long long seed) {

    //initialize thread index & stride for each thread 
    int tid = threadIdx.x;
    int stride = blockDim.x;
    int optId = blockIdx.x;         

 
    double call_sum = 0.0;
    double put_sum = 0.0;
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed + optId, tid, 0, &state);

    const OptionGPU p = trades_pointer[optId];


    const double drift     = (p.r - 0.5 * p.sigma * p.sigma) * p.T;
    const double diffusion = p.sigma * sqrt(p.T);

    for (int i = tid; i < num_simulations; i+=stride) {

        double Z = hiprand_normal_double(&state); 
        //stock price at expiration 
        double ST = p.S * exp(drift + diffusion * Z);
        //if strike price is greater than stock price at expiration, then payoff is zero
        //otherwise, payoff is stock price at expiration minus strike price (For call option)
        call_sum += fmax(ST - p.X, 0.0); 
        put_sum += fmax(p.X - ST, 0.0);
    }

    extern __shared__ double sdata[];  
    double* call_buf = sdata;
    double* put_buf  = sdata + blockDim.x;
    call_buf[tid] = call_sum;
    put_buf[tid]  = put_sum;
    __syncthreads(); 

    for (int offset = stride >> 1; offset > 0; offset >>= 1) {
    if (tid < offset)       
        {
                call_buf[tid] += call_buf[tid + offset];
                put_buf[tid]  += put_buf[tid + offset];
        }         
    __syncthreads();   

    }
    if (tid == 0) {                    
        // average of payoffs over number of simulations , then discounting back to present value
        trades_results[optId].call_price = exp(-p.r * p.T) * call_buf[0] / static_cast<double>(num_simulations);
        trades_results[optId].put_price = exp(-p.r * p.T) * put_buf[0] / static_cast<double>(num_simulations);
    }
}

//optional function to load trades from a CSV file
std::vector<optionParams> load_csv(const std::string& filename) {
    std::vector<optionParams> trades;
    std::ifstream in(filename);
    std::string line;

    if (!in.is_open()) {
        std::cerr << "Error opening file: " << filename << std::endl;
        return trades;
    }

    while (std::getline(in, line)) {
        std::istringstream ss(line);
        optionParams trade;
        char comma;
        ss >> trade.S >> comma >> trade.X >> comma >> trade.T >> comma >> trade.r >> comma >> trade.sigma;
        trades.push_back(trade);
    }
    return trades;
}

static void run_pricer(const std::vector<optionParams>&trades, const config& options) {

    //num threads
    int blockSize = 256;
    //number of blocks (in a grid) (one per option)
    int numBlocks = trades.size(); 
    size_t shmem    = 2 * blockSize * sizeof(double);

    std::vector<OptionGPU> gpuTrades(trades.size());
    for (size_t i=0; i<trades.size(); ++i) {
        gpuTrades[i] = { trades[i].S, trades[i].X, trades[i].T, trades[i].r, trades[i].sigma };
    }

    //allocate memory on GPU and copy data from CPU to GPU
    OptionGPU* d_trades;
    hipMallocManaged(&d_trades, gpuTrades.size()*sizeof(OptionGPU));
    hipMemcpy(d_trades, gpuTrades.data(),gpuTrades.size()*sizeof(OptionGPU),hipMemcpyHostToDevice);

    //allocate memory on GPU for results of each simulation 
    optionPrices *trades_results;
    hipMallocManaged(&trades_results, trades.size() * sizeof(optionPrices));
    
    monte_carlo_simulation <<<numBlocks, blockSize, shmem>>> (d_trades, trades_results, options.num_simulations,  /*seed=*/1234ULL);


    hipDeviceSynchronize(); 


    for (size_t i = 0; i < trades.size(); ++i) {
        const auto& opt = trades[i];
        std::cout << "Option " << i+1 << ": " << "S: " << opt.S << ", X: " << opt.X << ", Expiration date: " << 
        opt.expiration_date << ", T: " << opt.T << ", r: " << opt.r << ", sigma: " << opt.sigma << "\n";

        optionPrices eu_bs = calc_call(opt);

        std::cout << " Analytical call price: " << eu_bs.call_price << " | Monte Carlo call price: " << trades_results[i].call_price << "\n";
        std::cout << " Analytical put price: " << eu_bs.put_price << " | Monte Carlo put price: " << trades_results[i].put_price << "\n\n";

    }

    hipFree(d_trades);
    hipFree(trades_results);


}

config parse_cmd_args(int argc, char *argv[]) {
    config c;
    auto need = [&](int& i) -> char* {
        //value for flag not provided
        if (++i == argc)
            throw std::runtime_error("missing value for " + std::string(argv[i-1]));
        return argv[i];
    };

    for (int i = 1; i < argc; ++i) {
        std::string a = argv[i];

        if      (a == "--symbol")  c.ticker  = need(i);
        else if (a == "--paths")   c.num_simulations   = std::stol(need(i));
        else if (a == "--help") {
            std::cout << "Usage: ./pricer --symbol TICKER --paths N\n";
            std::cout << "if --paths is not specified, default is 1 million paths\n";
            std::exit(0);
        }
        else
            throw std::runtime_error("unknown flag: " + std::string(a));
    }
    if (c.ticker.empty())
        throw std::runtime_error("--symbol is required (try --help)");
    return c;
}

int main(int argc, char *argv[]) {

    config options;
    try {
        options = parse_cmd_args(argc, argv);
    }
    catch (const std::exception& e) {
        std::cerr << "Error parsing command line arguments: " << e.what() << std::endl;
        return 1;
    }

    double r = 0.0;
    try {
        r = fetch_risk_free_rate();
        std::cout << "Risk-free rate used (DGS3MO): " << r << "\n\n";
    } catch (const std::exception& e) {
        std::cerr << "Error in fetch risk free rate " << e.what() << std::endl;  
    }
    
    std::vector<optionParams> trades;
    try {
        trades = fetch_chain(options.ticker, r); 
    } catch (const std::exception& e) {
    std::cerr << "Error in fetch chain: " << e.what() << std::endl; 
    }

    auto start = std::chrono::steady_clock::now();
    run_pricer(trades, options);
    auto stop = std::chrono::steady_clock::now();
    std::chrono::duration<double> elapsed = stop - start;   


    double seconds = elapsed.count();          
    double throughput = options.num_simulations / seconds;          


    std::cout << "Simulation Summary\n"
     << "------------------\n" 
     << "Contracts processed: " << trades.size() << " options in " << seconds << " seconds \n"
    << "Paths per contract: " << options.num_simulations << "\n"
    << "Throughput: " << throughput << " paths/second\n";


    

    return 0;
}
